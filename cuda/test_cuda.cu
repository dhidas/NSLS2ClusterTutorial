#include <iostream>

int main (int argc, char* argv[])
{
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);

  if (ngpu <= 0) {
    std::cerr << "ERROR: no GPUs are available" << std::endl;
    return 1;
  }

  hipDeviceProp_t prop;

  for (int i = 0; i != ngpu; ++i) {
    hipGetDeviceProperties(&prop, i);
    std::cout << "GPU " << i << std::endl;
    std::cout << "  Device name: " << prop.name << std::endl;
    std::cout << "  Memory Clock Rate (KHz): " << prop.memoryClockRate << std::endl;
    std::cout << "  Memory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
    std::cout << "  Peak Memory Bandwidth (GB/s): " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 << std::endl;
    std::cout << std::endl;
  }

  return 0;
}
